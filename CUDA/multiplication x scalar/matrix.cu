#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__host__
void llenar(float *d_a, int tam) {
  int n = 10;
  for (int i = 0; i < tam; i++) {
    d_a[i] = n;
  }
}

void print(float *V, int tam){
  for (int i = 0; i < tam; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__

void mult_matKernel(float* h_a, float* h_b , int n) {
  int i = threadIdx.x + blockDim.x *blockIdx.x;
  if (i < n) {
    h_b[i] = h_a[i] * 2 ;
  }
}


int main(int argc, char const *argv[]) {
  int n = 100;

  float *h_a = (float*)malloc(n*sizeof(float));
  float *h_b = (float*)malloc(n*sizeof(float));


  hipError_t error = hipSuccess;
  float *d_a, *d_b;

  error = hipMalloc((void**)&d_a, n*sizeof(float));
  if (error != hipSuccess) {
    printf("Error al asignar espacio a d_a\n" );
    return 0;
  }

  error = hipMalloc((void**)&d_b, n*sizeof(float));
  if (error != hipSuccess) {
    printf("Error al asignar espacio a d_b\n" );
    return 0;
  }

  llenar(h_a, n);
  
 hipMemcpy(d_a, h_a, n*sizeof(float), hipMemcpyHostToDevice);
//print(d_a, n);
//print(h_a, n);
 dim3 dimGrid(ceil(n/10.0), 1, 1);
  dim3 dimBlock(10,1,1);

  mult_matKernel<<<dimGrid, dimBlock>>>(d_a, d_b, n);
hipDeviceSynchronize();
  hipMemcpy(h_b, d_b, n*sizeof(float), hipMemcpyDeviceToHost);


  print(h_b, n);

  free(h_a);
  free(h_b);
  hipFree(d_a);
  hipFree(d_b);

  return 0;
}
