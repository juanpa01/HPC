#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>

__host__
void llenar(float *h_a, int tam) {
  int n = 10;
  for (int i = 0; i < tam; i++) {
    h_a[i] = n;
  }
}

__host__
void print(float *V, int tam){
  for (int i = 0; i < tam; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__

void mult_mat(float* h_a, float* h_b , int n) {
  int i = threadIdx.x + blockDim.x *blockIdx.x;
  if (i < n) {
    h_b[i] = h_a[i] * 2 ;
  }
}


__global__
int main(int argc, char const *argv[]) {
  int n = 100;

  float *h_a = (float*)malloc(n*sizeof(float));
  float *h_b = (float*)malloc(n*sizeof(float));


  hipError_t error = hipSuccess;
  float *d_a, *d_b;

  error = hipMalloc((void**)&d_a, n*sizeof(float));
  if (error != hipSuccess) {
    printf("Error al asignar espacio a d_a\n", );
    return 1;
  }

  error = hipMalloc((void**)&d_b, n*sizeof(float));
  if (error != hipSuccess) {
    printf("Error al asignar espacio a d_a\n", );
    return 1;
  }

  llenar(h_a, n);

  hipMemcpy(d_a, h_a, n*sizeof(float), hipMemcpyHostToDevice);

  mult_mat<<<ceil(n/float(10)),10>>>(h_a, h_b, n);

  hipMemcpy(h_b, d_b, m*sizeof(float), hipMemcpyDeviceToHost);

  print(h_b, n);

  free(h_a);
  free(h_b);
  hipFree(d_a);
  hipFree(d_b);

  return 0;
}
